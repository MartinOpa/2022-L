#include "hip/hip_runtime.h"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cuda_img.h"

__global__ void kernel_draw_circle( CudaImg t_color_cuda_img, int2 t_pos, int t_r, uchar3 t_color )
{

	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_color_cuda_img.m_size.y ) return;
	if ( l_x >= t_color_cuda_img.m_size.x ) return;

	int l_dx = l_x - t_pos.x;
	int l_dy = l_y - t_pos.y;
	int l_r = sqrtf( l_dx * l_dx + l_dy * l_dy );

	t_color_cuda_img.m_p_uchar4[ l_y * t_color_cuda_img.m_size.x + l_x ] =
		{ t_color.x, t_color.y, t_color.z, ( unsigned char ) ( 1 *  (l_r < t_r))};
}

void cu_draw_circle( CudaImg t_color_cuda_img, int2 t_pos, int t_r, uchar3 t_color )
{
	hipError_t l_cerr;

	int l_block_size = 32;
	dim3 l_blocks( ( t_color_cuda_img.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_color_cuda_img.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_draw_circle<<< l_blocks, l_threads >>>( t_color_cuda_img, t_pos, t_r, t_color );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}

__global__ void kernel_insertimage( CudaImg t_big_cuda_pic, CudaImg t_small_cuda_pic, int2 t_position )
{
	int l_y = blockDim.y * blockIdx.y + threadIdx.y;
	int l_x = blockDim.x * blockIdx.x + threadIdx.x;
	if ( l_y >= t_small_cuda_pic.m_size.y ) return;
	if ( l_x >= t_small_cuda_pic.m_size.x ) return;
	int l_by = l_y + t_position.y;
	int l_bx = l_x + t_position.x;
	if ( l_by >= t_big_cuda_pic.m_size.y || l_by < 0 ) return;
	if ( l_bx >= t_big_cuda_pic.m_size.x || l_bx < 0 ) return;

	uchar4 l_fg_bgra = t_small_cuda_pic.m_p_uchar4[ l_y * t_small_cuda_pic.m_size.x + l_x ];
	uchar3 l_bg_bgr = t_big_cuda_pic.m_p_uchar3[ l_by * t_big_cuda_pic.m_size.x + l_bx ];
	uchar3 l_bgr = { 0, 0, 0 };

	l_bgr.x = l_fg_bgra.x * l_fg_bgra.w / 255 + l_bg_bgr.x * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.y = l_fg_bgra.y * l_fg_bgra.w / 255 + l_bg_bgr.y * ( 255 - l_fg_bgra.w ) / 255;
	l_bgr.z = l_fg_bgra.z * l_fg_bgra.w / 255 + l_bg_bgr.z * ( 255 - l_fg_bgra.w ) / 255;

	t_big_cuda_pic.m_p_uchar3[ l_by * t_big_cuda_pic.m_size.x + l_bx ] = l_bgr;
}

void cu_insertimage( CudaImg t_big_cuda_pic, CudaImg t_small_cuda_pic, int2 t_position )
{
	hipError_t l_cerr;

	int l_block_size = 32;
	dim3 l_blocks( ( t_small_cuda_pic.m_size.x + l_block_size - 1 ) / l_block_size,
			       ( t_small_cuda_pic.m_size.y + l_block_size - 1 ) / l_block_size );
	dim3 l_threads( l_block_size, l_block_size );
	kernel_insertimage<<< l_blocks, l_threads >>>( t_big_cuda_pic, t_small_cuda_pic, t_position );

	if ( ( l_cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( l_cerr ) );

	hipDeviceSynchronize();
}


